#include "hip/hip_runtime.h"
#include "CUDAFuncs.h"

#define ANCILLA_PHOTONS 6
#define ANCILLA_MODES 8

__constant__ double dev_factorial[ ANCILLA_PHOTONS + 2 + 1 ];
__constant__ double dev_U[ 2 * (ANCILLA_MODES + 4) * (ANCILLA_MODES + 4) ];


__global__ void setEachUTerm(){


}


void CUDAOffloader::allocateResources(){

    int count;

    hipGetDeviceCount( &count );

    hipDeviceProp_t prop[count];

    for(int i=0;i<count;i++) hipGetDeviceProperties( &prop[i],i );

    int UStorageSize = ( ANCILLA_MODES + 4 ) * ( ANCILLA_MODES + 4 ) * 16;
    int factorialStorageSize = ( ANCILLA_PHOTONS + 3 ) * 8;

    int spaceAvail[ count ];

    for(int i=0;i<count;i++) spaceAvail[i] = prop[i].totalGlobalMem;

    for(int i=0;i<count;i++) spaceAvail[i] -= ( UStorageSize + factorialStorageSize );

    int maxTerms[ count ];

    for(int i=0;i<count;i++){

        maxTerms[i] = 0;

        int spaceTaken = 0;

        while( spaceTaken < spaceAvail[i] ){

            spaceTaken = maxTerms[i] * 4 * ( 4 + ANCILLA_MODES );
            spaceTaken += maxTerms[i] * 4 * ( 2 + ANCILLA_PHOTONS );
            spaceTaken += maxTerms[i] * 16;

            maxTerms[i]++;

        }

        maxTerms[i]--;

    }

    maxTerms[1] = 0;

    std::cout << "Terms that can be done on GTX at once: " << maxTerms[0] << std::endl;
    std::cout << "Terms that can be done on NVS at once: " << maxTerms[1] << std::endl;
    std::cout << "Total terms: " << numberOfTerms << std::endl;

    blocksPerGrid.resize(count);
    threadsPerBlock.resize(count);
    termsPerIteration.resize(count);

    for(int i=0;i<count;i++) threadsPerBlock.at(i) = prop[i].maxThreadsPerBlock;

    for(int i=0;i<count;i++) blocksPerGrid.at(i) = maxTerms[i] / threadsPerBlock[i];

    for(int i=0;i<count;i++) termsPerIteration.at(i) = blocksPerGrid.at(i) * threadsPerBlock.at(i);

    for(int i=0;i<count;i++) assert( termsPerIteration.at(i) <= maxTerms[i] );

    totalTermsPerIteration = 0;
    for(int i=0;i<count;i++) totalTermsPerIteration += termsPerIteration.at(i);

    iterations = ( numberOfTerms + totalTermsPerIteration - 1 ) / totalTermsPerIteration;

    for(int i=0;i<count;i++) std::cout << "Blocks Per Grid: " << blocksPerGrid.at(i) << std::endl;
    for(int i=0;i<count;i++) std::cout << "Threads Per Block: " << threadsPerBlock.at(i) << std::endl;
    for(int i=0;i<count;i++) std::cout << "Terms Per Iteration: " << termsPerIteration.at(i) << std::endl;

    std::cout << "Total Terms Per Iteration: " << totalTermsPerIteration << std::endl;
    std::cout << "Iterations: " << iterations << std::endl;

    numbGPUs = count;

    nPrimeSub = new int[ totalTermsPerIteration * ( 4 + ANCILLA_MODES ) ];
    mPrimeSub = new int[ totalTermsPerIteration * ( 2 + ANCILLA_PHOTONS ) ];

    return;

}

void CUDAOffloader::sendUToGPU(Eigen::MatrixXcd& U){

    double UArr[ 2 * (ANCILLA_MODES + 4) * (ANCILLA_MODES + 4) ];

    int k=0;

    for(int j=0;j<ANCILLA_MODES+4;j++) for(int i=0;i<ANCILLA_MODES+4;i++){

        UArr[k] = std::real( U(i,j) );
        k++;

        UArr[k] = std::imag( U(i,j) );
        k++;

    }

    hipMemcpyToSymbol(HIP_SYMBOL( dev_U),UArr, 2 * (ANCILLA_MODES + 4) * (ANCILLA_MODES + 4) * sizeof(double) );

    return;

}

void CUDAOffloader::setSubNPrimeMPrime(std::vector< std::vector<int> >& nPrime,std::vector< std::vector<int> >& mPrime){

    int kn = 0;
    int km = 0;

    int subWall = 0;

    while( subWall < totalTermsPerIteration ){

        do{

            if( subWall >= totalTermsPerIteration ){

                subIndex--;
                break;

            }

            for(int i=0;i<nPrime[ subIndex ].size();i++){

                nPrimeSub[ kn ] = nPrime[ subIndex ][i];

                kn++;

            }

            for(int i=0;i<mPrime[ subIndex ].size();i++){

                mPrimeSub[ km ] = mPrime[ subIndex ][i];

                km++;

            }

            subWall++;

        } while( std::next_permutation( mPrime[ subIndex ].begin(), mPrime[ subIndex ].end()  ) );

        subIndex++;

    }

    return;

}


double CUDAOffloader::setMutualEntropy(std::vector< std::vector<int> >& nPrime,std::vector< std::vector<int> >& mPrime){

    subIndex = 0;

    setSubNPrimeMPrime(nPrime,mPrime);

    setEachUTerm<<<10,10>>>();

    std::cout << "CUDA Errors: " << hipGetErrorString( hipGetLastError() ) << std::endl;

    return 1.0;

}


CUDAOffloader::CUDAOffloader(){


}


void CUDAOffloader::setGPUDevice(int deviceNumb){

    hipSetDevice(deviceNumb);

    return;

}

void CUDAOffloader::sendFactorialToGPU(std::vector<double>& factorial){

    assert( ANCILLA_PHOTONS + 2 + 1 == factorial.size() );

    double factorialArr[factorial.size()];

    for(int i=0;i<factorial.size();i++) factorialArr[i] = factorial.at(i);

    hipMemcpyToSymbol(HIP_SYMBOL( dev_factorial),factorialArr, factorial.size() * sizeof(double) );

    return;

}


void CUDAOffloader::queryGPUDevices(){

    int count;

    hipGetDeviceCount( &count );

    std::cout << "Number of devices: " << count << std::endl << std::endl;

    for(int i=0;i<count;i++){

        hipDeviceProp_t prop;

        hipGetDeviceProperties( &prop,i );

        std::cout << "Device No. " << i << ": " << std::endl;
        std::cout << "\t" << prop.name << std::endl;
        if(prop.integrated) std::cout << "\tIntegrated GPU" << std::endl;
        else std::cout << "\tNon-integrated GPU" << std::endl;
        std::cout << "\t" << "Device compute capability: " << prop.major << "." << prop.minor << " (1.3 or higher supports double-precision math)" << std::endl;
        std::cout << "\t" << prop.totalGlobalMem << " bytes of global memory" << std::endl;
        std::cout << "\t" << prop.sharedMemPerBlock << " bytes of shared memory for a single block" << std::endl;
        std::cout << "\t" << prop.regsPerBlock << " registers (32 bit) available per block" << std::endl;
        std::cout << "\t" << prop.warpSize << " threads in a warp" << std::endl;
        std::cout << "\t" << prop.memPitch << " bytes maximum pitch allowed for memory copies" << std::endl;
        std::cout << "\t" << prop.maxThreadsPerBlock << " maximum number of threads that a block may contain" << std::endl;
        std::cout << "\t" << prop.maxThreadsDim[0] << " maximum number of threads along X" << std::endl;
        std::cout << "\t" << prop.totalConstMem << " amount of available constant memory" << std::endl;
        if(prop.deviceOverlap) std::cout << "\t" << "Device can simultaneously perform hipMemcpy() and a kernel execution" << std::endl;
        else std::cout << "\t" << "Device cannot simultaneously perform hipMemcpy() and a kernel execution" << std::endl;
        std::cout << "\t" << prop.multiProcessorCount << " multiprocessors on the device" << std::endl;
        if(prop.kernelExecTimeoutEnabled) std::cout << "\tRuntime limit for kernels on this device is enabled" << std::endl;
        else std::cout << "\tRuntime limit for kernels on this device is disabled" << std::endl;

        std::cout << "\t" << "Compute mode: " << prop.computeMode << std::endl;
        std::cout << "\t" << "Concurrent Kernels: " << prop.concurrentKernels << std::endl;

        prop.computeMode = 1;

        std::cout << std::endl;

    }

    return;

}
