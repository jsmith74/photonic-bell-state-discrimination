#include "hip/hip_runtime.h"
#include "CUDAFuncs.h"

#define ANCILLA_PHOTONS 6
#define ANCILLA_MODES 8
#define HILBERT_SPACE_DIMENSION 75582

#define TERMS_BUFFER 5

// REMEMBER TO DELETE DYNAMIC MEMORY DECLARED BY nPrimeStarter and mPrimeStarter AT THE END OF THE OPTIMIZATION ROUTINE

__constant__ double dev_factorial[ ANCILLA_PHOTONS + 2 + 1 ];
__constant__ double dev_U[ 2 * (ANCILLA_MODES + 4) * (ANCILLA_MODES + 4) ];
__constant__ int dev_termIntervals;

__device__ bool next_permutation(int* __first, int* __last);

//thrust::complex<double>* dev_UTerms;

__global__ void kernel(int* dev_nPrime,int* dev_mPrime){

    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    int term = 0;

    while(term < dev_termIntervals){

        do{

            // write code to keep track of which terms of dev_UTerms belong where, factor
            // this into the space allocation function

            // dev_nPrime[ i + tid * (4 + ANCILLA_MODES) ];
            // dev_mPrime[ i + tid * (2 + ANCILLA_PHOTONS) ];

            term++;

            if(term >= dev_termIntervals) break;

        } while( next_permutation( &dev_mPrime[ tid * (ANCILLA_PHOTONS + 2) ] , &dev_mPrime[ (tid + 1) * (ANCILLA_PHOTONS + 2) ] ) );

        // iterate dev_nPrime and generate new corresponding dev_mPrime in initialized position

    }

}

double CUDAOffloader::setMutualEntropy(){

    std::cout << "Begin..." << std::endl;

    int* dev_nPrime;    int* dev_mPrime;

    hipMalloc( (void**)&dev_nPrime, numberOfThreads * ( 4 + ANCILLA_MODES ) * sizeof(int) );

    hipMalloc( (void**)&dev_mPrime, numberOfThreads * ( 2 + ANCILLA_PHOTONS ) * sizeof(int) );

    hipMemcpy( dev_nPrime, nPrimeStarter, numberOfThreads * ( 4 + ANCILLA_MODES ) * sizeof(int), hipMemcpyHostToDevice );

    hipMemcpy( dev_mPrime, mPrimeStarter, numberOfThreads * ( 2 + ANCILLA_PHOTONS ) * sizeof(int), hipMemcpyHostToDevice );

    kernel<<<blocksPerGrid,threadsPerBlock>>>(dev_nPrime,dev_mPrime);

    hipFree( dev_nPrime );

    hipFree( dev_mPrime );

    std::cout << "End." << std::endl;

    std::cout << "CUDA Errors: " << hipGetErrorString( hipGetLastError() ) << std::endl;

    return 1.0;

}

void CUDAOffloader::initializeStartingNPrimeMPrime(std::vector< std::vector<int> >& nPrime,std::vector< std::vector<int> >& mPrime){

    nPrimeStarter = new int[ numberOfThreads * (4 + ANCILLA_MODES) ];
    mPrimeStarter = new int[ numberOfThreads * (2 + ANCILLA_PHOTONS) ];

    gccCompiledFunctions.initializeStartingNPrimeMPrime(nPrime,mPrime,nPrimeStarter,mPrimeStarter,numberOfThreads,termIntervals);

    return;

}

void CUDAOffloader::allocateResources(){

    int count;

    hipGetDeviceCount( &count );

    assert( count > 0 );

    hipDeviceProp_t prop;

    hipGetDeviceProperties( &prop,0 );

    int spaceAvail = prop.totalGlobalMem;

    int UStorageSize = ( ANCILLA_MODES + 4 ) * ( ANCILLA_MODES + 4 ) * 16;
    int factorialStorageSize = ( ANCILLA_PHOTONS + 3 ) * 8;
    int UTermStorageSize = 16 * HILBERT_SPACE_DIMENSION;

    spaceAvail -= UStorageSize;
    spaceAvail -= factorialStorageSize;
    spaceAvail -= UTermStorageSize;

    std::cout << "Space available on GPU: " << spaceAvail << " bytes" <<std::endl << std::endl;

    numberOfThreads = 0;
    int spaceTaken = 0;

    while( spaceTaken < spaceAvail ){

        numberOfThreads++;

        spaceTaken = 4 * numberOfThreads * ( 2 + 4 + ANCILLA_PHOTONS + ANCILLA_MODES );

        spaceTaken += 16 * numberOfThreads;

    }

    numberOfThreads--;

    threadsPerBlock = 1024;

    std::cout << "Number of total terms: " << numberOfTerms << std::endl;
    std::cout << "Max Number of threads: " << numberOfThreads << std::endl;
    std::cout << "Space used on GPU for this number: " << 4 * numberOfThreads * ( 2 + 4 + ANCILLA_PHOTONS + ANCILLA_MODES + 4 ) << " bytes" << std::endl;

    termIntervals = ( numberOfTerms + numberOfThreads - 1 ) / numberOfThreads;
    termIntervals += TERMS_BUFFER;

    std::cout << "The Minimum Number of Terms that need to be evaluated in at least one interval: " << termIntervals << std::endl;

    numberOfThreads = ( numberOfTerms + termIntervals - 1 ) / termIntervals;

    std::cout << "Adjusted number of threads: " << numberOfThreads << std::endl;
    std::cout << "Number of total terms covered if each thread does " << termIntervals << " terms: "  << numberOfThreads * termIntervals << std::endl;

    while( numberOfThreads % threadsPerBlock != 0 ) threadsPerBlock--;

    blocksPerGrid = numberOfThreads / threadsPerBlock;

    std::cout << "Adjusted threads per block: " << threadsPerBlock << std::endl;
    std::cout << "Adjusted blocks per grid: " << blocksPerGrid << std::endl;

    std::cout << "Adjusted Space used on GPU: " << 4 * numberOfThreads * ( 2 + 4 + ANCILLA_PHOTONS + ANCILLA_MODES + 4 ) << " bytes" << std::endl;

    assert( threadsPerBlock * blocksPerGrid == numberOfThreads );

    hipMemcpyToSymbol(HIP_SYMBOL( dev_termIntervals),&termIntervals, sizeof(int) );

    return;

}

void CUDAOffloader::sendUToGPU(Eigen::MatrixXcd& U){

    double UArr[ 2 * (ANCILLA_MODES + 4) * (ANCILLA_MODES + 4) ];

    int k=0;

    for(int j=0;j<ANCILLA_MODES+4;j++) for(int i=0;i<ANCILLA_MODES+4;i++){

        UArr[k] = std::real( U(i,j) );
        k++;

        UArr[k] = std::imag( U(i,j) );
        k++;

    }

    hipMemcpyToSymbol(HIP_SYMBOL( dev_U),UArr, 2 * (ANCILLA_MODES + 4) * (ANCILLA_MODES + 4) * sizeof(double) );

    return;

}

CUDAOffloader::CUDAOffloader(){


}


void CUDAOffloader::setGPUDevice(int deviceNumb){

    hipSetDevice(deviceNumb);

    return;

}

void CUDAOffloader::sendFactorialToGPU(std::vector<double>& factorial){

    assert( ANCILLA_PHOTONS + 2 + 1 == factorial.size() );

    double factorialArr[factorial.size()];

    for(int i=0;i<factorial.size();i++) factorialArr[i] = factorial.at(i);

    hipMemcpyToSymbol(HIP_SYMBOL( dev_factorial),factorialArr, factorial.size() * sizeof(double) );

    return;

}


void CUDAOffloader::queryGPUDevices(){

    int count;

    hipGetDeviceCount( &count );

    std::cout << "Number of devices: " << count << std::endl << std::endl;

    for(int i=0;i<count;i++){

        hipDeviceProp_t prop;

        hipGetDeviceProperties( &prop,i );

        std::cout << "Device No. " << i << ": " << std::endl;
        std::cout << "\t" << prop.name << std::endl;
        if(prop.integrated) std::cout << "\tIntegrated GPU" << std::endl;
        else std::cout << "\tNon-integrated GPU" << std::endl;
        std::cout << "\t" << "Device compute capability: " << prop.major << "." << prop.minor << " (1.3 or higher supports double-precision math)" << std::endl;
        std::cout << "\t" << prop.totalGlobalMem << " bytes of global memory" << std::endl;
        std::cout << "\t" << prop.sharedMemPerBlock << " bytes of shared memory for a single block" << std::endl;
        std::cout << "\t" << prop.regsPerBlock << " registers (32 bit) available per block" << std::endl;
        std::cout << "\t" << prop.warpSize << " threads in a warp" << std::endl;
        std::cout << "\t" << prop.memPitch << " bytes maximum pitch allowed for memory copies" << std::endl;
        std::cout << "\t" << prop.maxThreadsPerBlock << " maximum number of threads that a block may contain" << std::endl;
        std::cout << "\t" << prop.maxThreadsDim[0] << " maximum number of threads along X" << std::endl;
        std::cout << "\t" << prop.totalConstMem << " amount of available constant memory" << std::endl;
        if(prop.deviceOverlap) std::cout << "\t" << "Device can simultaneously perform hipMemcpy() and a kernel execution" << std::endl;
        else std::cout << "\t" << "Device cannot simultaneously perform hipMemcpy() and a kernel execution" << std::endl;
        std::cout << "\t" << prop.multiProcessorCount << " multiprocessors on the device" << std::endl;
        if(prop.kernelExecTimeoutEnabled) std::cout << "\tRuntime limit for kernels on this device is enabled" << std::endl;
        else std::cout << "\tRuntime limit for kernels on this device is disabled" << std::endl;

        std::cout << "\t" << "Compute mode: " << prop.computeMode << std::endl;
        std::cout << "\t" << "Concurrent Kernels: " << prop.concurrentKernels << std::endl;

        prop.computeMode = 1;

        std::cout << std::endl;

    }

    return;

}


__device__ inline void iter_swap(int* __a, int* __b) {
  int __tmp = *__a;
  *__a = *__b;
  *__b = __tmp;
}


__device__ void reverse(int* __first, int* __last) {

  while (true)
    if (__first == __last || __first == --__last)
      return;
    else{
      iter_swap(__first++, __last);
    }
}


__device__ bool next_permutation(int* __first, int* __last) {

  if (__first == __last)
    return false;
  int* __i = __first;
  ++__i;
  if (__i == __last)
    return false;
  __i = __last;
  --__i;

  for(;;) {
    int* __ii = __i;
    --__i;
    if (*__i < *__ii) {
      int* __j = __last;
      while (!(*__i < *--__j))
        {}
    iter_swap(__i, __j);
      reverse(__ii, __last);
      return true;
    }
    if (__i == __first) {
      reverse(__first, __last);
      return false;
    }
  }

}
