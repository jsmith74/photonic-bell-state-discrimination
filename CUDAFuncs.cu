#include "hip/hip_runtime.h"
#include "CUDAFuncs.h"

#define ANCILLA_PHOTONS 6
#define ANCILLA_MODES 8



__global__ void setEachUTerm(){

    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    int test = 33;
    int test2 = 52;

    int test3 = test + test2;

}

void CUDAOffloader::sendUtoGPU(Eigen::MatrixXcd& U){

    return;

}

double CUDAOffloader::setMutualEntropy(){

    const int threadsPerBlock = 1014;
    const int blocksPerGrid = ( numberOfTerms + threadsPerBlock - 1 ) / threadsPerBlock;;

    std::cout << blocksPerGrid << "\t" << numberOfTerms << std::endl;
    std::cout << blocksPerGrid * threadsPerBlock << std::endl;

    setEachUTerm<<<blocksPerGrid,threadsPerBlock>>>();

    return 1.0;

}


CUDAOffloader::CUDAOffloader(){


}


void CUDAOffloader::setGPUDevice(int deviceNumb){

    hipSetDevice(deviceNumb);

    return;

}

void CUDAOffloader::queryGPUDevices(){

    int count;

    hipGetDeviceCount( &count );

    std::cout << "Number of devices: " << count << std::endl << std::endl;

    for(int i=0;i<count;i++){

        hipDeviceProp_t prop;

        hipGetDeviceProperties( &prop,i );

        std::cout << "Device No. " << i << ": " << std::endl;
        std::cout << "\t" << prop.name << std::endl;
        if(prop.integrated) std::cout << "\tIntegrated GPU" << std::endl;
        else std::cout << "\tNon-integrated GPU" << std::endl;
        std::cout << "\t" << "Device compute capability: " << prop.major << "." << prop.minor << " (1.3 or higher supports double-precision math)" << std::endl;
        std::cout << "\t" << prop.totalGlobalMem << " bytes of global memory" << std::endl;
        std::cout << "\t" << prop.sharedMemPerBlock << " bytes of shared memory for a single block" << std::endl;
        std::cout << "\t" << prop.regsPerBlock << " registers (32 bit) available per block" << std::endl;
        std::cout << "\t" << prop.warpSize << " threads in a warp" << std::endl;
        std::cout << "\t" << prop.memPitch << " bytes maximum pitch allowed for memory copies" << std::endl;
        std::cout << "\t" << prop.maxThreadsPerBlock << " maximum number of threads that a block may contain" << std::endl;
        std::cout << "\t" << prop.maxThreadsDim[0] << " maximum number of threads along X" << std::endl;
        std::cout << "\t" << prop.totalConstMem << " amount of available constant memory" << std::endl;
        if(prop.deviceOverlap) std::cout << "\t" << "Device can simultaneously perform hipMemcpy() and a kernel execution" << std::endl;
        else std::cout << "\t" << "Device cannot simultaneously perform hipMemcpy() and a kernel execution" << std::endl;
        std::cout << "\t" << prop.multiProcessorCount << " multiprocessors on the device" << std::endl;
        if(prop.kernelExecTimeoutEnabled) std::cout << "\tRuntime limit for kernels on this device is enabled" << std::endl;
        else std::cout << "\tRuntime limit for kernels on this device is disabled" << std::endl;

        std::cout << std::endl;

    }

    return;

}
