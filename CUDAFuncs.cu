#include "hip/hip_runtime.h"
#include "CUDAFuncs.h"

#define ANCILLA_PHOTONS 6
#define ANCILLA_MODES 8
#define HILBERT_SPACE_DIMENSION 75582

#define TERMS_BUFFER 20

// REMEMBER TO DELETE DYNAMIC MEMORY DECLARED BY nPrimeStarter and mPrimeStarter and reduceGridStart and reduceGridEnd AT THE END OF THE OPTIMIZATION ROUTINE

__constant__ double dev_factorial[ ANCILLA_PHOTONS + 2 + 1 ];
__constant__ double dev_U[ 2 * (ANCILLA_MODES + 4) * (ANCILLA_MODES + 4) ];
__constant__ int dev_termIntervals;
__constant__ int dev_reduceGridSize;

__device__ bool next_permutation(int* __first, int* __last);
__device__ bool iterateNPrime(int* __begin,int* __end);
__device__ void setMPrime( int* __nBegin, int* __mBegin );

__device__ thrust::complex<double> Uel(int i,int j){

    thrust::complex<double> I(0.0,1.0);

    return dev_U[ 2 * ( i + j * ( ANCILLA_MODES + 4 ) ) ] + dev_U[ 2 * ( i + j * ( ANCILLA_MODES + 4 ) ) + 1 ] * I;

}

__global__ void kernel(int* dev_nPrime,int* dev_mPrime,thrust::complex<double>* dev_UTermBegin,thrust::complex<double>* dev_UTermEnd,double* dev_HXYMid){

    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    int term = 0;

    dev_UTermEnd[ tid ] = 0;
    dev_UTermEnd[ tid + 1 ] = 0;
    dev_UTermEnd[ tid + 2 ] = 0;
    dev_UTermEnd[ tid + 3 ] = 0;

    dev_HXYMid[ tid ] = 0;

    bool start = true;

    while(term < dev_termIntervals){

        do{

            thrust::complex<double> UProdTemp(1.0,0.0);

            for(int i=0;i<ANCILLA_PHOTONS;i++) UProdTemp *= Uel( i, dev_mPrime[ tid * (ANCILLA_PHOTONS + 2) + i ] );

            dev_UTermEnd[ tid ] += UProdTemp * (
                                              Uel( ANCILLA_MODES , dev_mPrime[ tid * (ANCILLA_PHOTONS + 2) + ANCILLA_PHOTONS ] ) * Uel( ANCILLA_MODES + 2, dev_mPrime[ tid * (ANCILLA_PHOTONS + 2) + ANCILLA_PHOTONS + 1 ] )
                                            + Uel( ANCILLA_MODES + 1, dev_mPrime[ tid * (ANCILLA_PHOTONS + 2) + ANCILLA_PHOTONS ] ) * Uel( ANCILLA_MODES + 3, dev_mPrime[ tid * (ANCILLA_PHOTONS + 2) + ANCILLA_PHOTONS + 1 ] )
                                         );

            dev_UTermEnd[ tid + 1 ] += UProdTemp * (
                                              Uel( ANCILLA_MODES , dev_mPrime[ tid * (ANCILLA_PHOTONS + 2) + ANCILLA_PHOTONS ] ) * Uel( ANCILLA_MODES + 3, dev_mPrime[ tid * (ANCILLA_PHOTONS + 2) + ANCILLA_PHOTONS + 1 ] )
                                            + Uel( ANCILLA_MODES + 1, dev_mPrime[ tid * (ANCILLA_PHOTONS + 2) + ANCILLA_PHOTONS ] ) * Uel( ANCILLA_MODES + 2, dev_mPrime[ tid * (ANCILLA_PHOTONS + 2) + ANCILLA_PHOTONS + 1 ] )
                                         );

            dev_UTermEnd[ tid + 2 ] += UProdTemp * (
                                              Uel( ANCILLA_MODES , dev_mPrime[ tid * (ANCILLA_PHOTONS + 2) + ANCILLA_PHOTONS ] ) * Uel( ANCILLA_MODES + 2, dev_mPrime[ tid * (ANCILLA_PHOTONS + 2) + ANCILLA_PHOTONS + 1 ] )
                                            - Uel( ANCILLA_MODES + 1, dev_mPrime[ tid * (ANCILLA_PHOTONS + 2) + ANCILLA_PHOTONS ] ) * Uel( ANCILLA_MODES + 3, dev_mPrime[ tid * (ANCILLA_PHOTONS + 2) + ANCILLA_PHOTONS + 1 ] )
                                         );

            dev_UTermEnd[ tid + 3 ] += UProdTemp * (
                                              Uel( ANCILLA_MODES , dev_mPrime[ tid * (ANCILLA_PHOTONS + 2) + ANCILLA_PHOTONS ] ) * Uel( ANCILLA_MODES + 3, dev_mPrime[ tid * (ANCILLA_PHOTONS + 2) + ANCILLA_PHOTONS + 1 ] )
                                            - Uel( ANCILLA_MODES + 1, dev_mPrime[ tid * (ANCILLA_PHOTONS + 2) + ANCILLA_PHOTONS ] ) * Uel( ANCILLA_MODES + 2, dev_mPrime[ tid * (ANCILLA_PHOTONS + 2) + ANCILLA_PHOTONS + 1 ] )
                                         );

            term++;

            if(term >= dev_termIntervals) break;

        } while( next_permutation( &dev_mPrime[ tid * (ANCILLA_PHOTONS + 2) ] , &dev_mPrime[ (tid + 1) * (ANCILLA_PHOTONS + 2) ] ) );

        dev_UTermEnd[ tid ] *= 0.7071067811865475;
        dev_UTermEnd[ tid + 1 ] *= 0.7071067811865475;
        dev_UTermEnd[ tid + 2 ] *= 0.7071067811865475;
        dev_UTermEnd[ tid + 3 ] *= 0.7071067811865475;

        for(int p=0;p<ANCILLA_MODES + 4;p++){

            dev_UTermEnd[ tid ] *= sqrt( dev_factorial[ dev_nPrime[ tid * (4 + ANCILLA_MODES) + p ] ] );
            dev_UTermEnd[ tid + 1 ] *= sqrt( dev_factorial[ dev_nPrime[ tid * (4 + ANCILLA_MODES) + p ] ] );
            dev_UTermEnd[ tid + 2 ] *= sqrt( dev_factorial[ dev_nPrime[ tid * (4 + ANCILLA_MODES) + p ] ] );
            dev_UTermEnd[ tid + 3 ] *= sqrt( dev_factorial[ dev_nPrime[ tid * (4 + ANCILLA_MODES) + p ] ] );

        }

        if(start){

            dev_UTermBegin[ tid ] = dev_UTermEnd[ tid ];
            dev_UTermBegin[ tid + 1 ] = dev_UTermEnd[ tid + 1 ];
            dev_UTermBegin[ tid + 2 ] = dev_UTermEnd[ tid + 2 ];
            dev_UTermBegin[ tid + 3 ] = dev_UTermEnd[ tid + 3 ];

            dev_UTermEnd[ tid ] = 0;
            dev_UTermEnd[ tid + 1 ] = 0;
            dev_UTermEnd[ tid + 2 ] = 0;
            dev_UTermEnd[ tid + 3 ] = 0;

            start = false;

        }

        else if(term >= dev_termIntervals) break;

        else{

            dev_HXYMid[ tid ] += thrust::norm( dev_UTermEnd[ tid ] ) * log2( ( thrust::norm( dev_UTermEnd[ tid ] ) + thrust::norm( dev_UTermEnd[ tid + 1 ] ) + thrust::norm( dev_UTermEnd[ tid + 2 ] ) + thrust::norm( dev_UTermEnd[ tid + 3 ] ) ) / thrust::norm( dev_UTermEnd[ tid ] ) );
            dev_HXYMid[ tid ] += thrust::norm( dev_UTermEnd[ tid + 1 ] ) * log2( ( thrust::norm( dev_UTermEnd[ tid ] ) + thrust::norm( dev_UTermEnd[ tid + 1 ] ) + thrust::norm( dev_UTermEnd[ tid + 2 ] ) + thrust::norm( dev_UTermEnd[ tid + 3 ] ) ) / thrust::norm( dev_UTermEnd[ tid + 1 ] ) );
            dev_HXYMid[ tid ] += thrust::norm( dev_UTermEnd[ tid + 2 ] ) * log2( ( thrust::norm( dev_UTermEnd[ tid ] ) + thrust::norm( dev_UTermEnd[ tid + 1 ] ) + thrust::norm( dev_UTermEnd[ tid + 2 ] ) + thrust::norm( dev_UTermEnd[ tid + 3 ] ) ) / thrust::norm( dev_UTermEnd[ tid + 2 ] ) );
            dev_HXYMid[ tid ] += thrust::norm( dev_UTermEnd[ tid + 3 ] ) * log2( ( thrust::norm( dev_UTermEnd[ tid ] ) + thrust::norm( dev_UTermEnd[ tid + 1 ] ) + thrust::norm( dev_UTermEnd[ tid + 2 ] ) + thrust::norm( dev_UTermEnd[ tid + 3 ] ) ) / thrust::norm( dev_UTermEnd[ tid + 3 ] ) );

            dev_UTermEnd[ tid ] = 0;
            dev_UTermEnd[ tid + 1 ] = 0;
            dev_UTermEnd[ tid + 2 ] = 0;
            dev_UTermEnd[ tid + 3 ] = 0;

        }

        if( tid == gridDim.x * blockDim.x - 1 && dev_nPrime[ ( tid + 1 ) * (4 + ANCILLA_MODES) - 1 ] == 2 + ANCILLA_PHOTONS ) break;

        iterateNPrime( &dev_nPrime[ tid * (4 + ANCILLA_MODES) ], &dev_nPrime[ (tid+1) * (4 + ANCILLA_MODES) ] );

        setMPrime( &dev_nPrime[ tid * (4 + ANCILLA_MODES) ], &dev_mPrime[ tid * (ANCILLA_PHOTONS + 2) ]);

    }

}

__global__ void reduce(thrust::complex<double>* dev_UTermBegin,thrust::complex<double>* dev_UTermEnd,double* dev_HXYMid,int* dev_reduceGridStart,int* dev_reduceGridEnd){

    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if( tid < dev_reduceGridSize ){

        // BUT IN CODE HERE TO COMBINE THE BLOCKS TOGETHER

    }

}

void CUDAOffloader::setReduceGrid(std::vector< std::vector<int> >& nPrime,std::vector< std::vector<int> >& mPrime){

    Eigen::MatrixXi tempReduceGrid;

    gccCompiledFunctions.setReduceGrid(nPrime,mPrime,termIntervals,tempReduceGrid);

    reduceGridSize = tempReduceGrid.rows();

    reduceGridStart = new int[ reduceGridSize ];
    reduceGridEnd = new int[ reduceGridSize ];

    for(int i=0;i<reduceGridSize;i++){

        reduceGridStart[i] = tempReduceGrid(i,0);
        reduceGridEnd[i] = tempReduceGrid(i,1);

    }

    tempReduceGrid.resize(0,0);

    hipMemcpyToSymbol(HIP_SYMBOL( dev_reduceGridSize),&reduceGridSize, sizeof(int) );

    return;

}

double CUDAOffloader::setMutualEntropy(){

    std::cout << "Begin..." << std::endl;

    int* dev_nPrime;    int* dev_mPrime;

    thrust::complex<double>* dev_UTermBegin;
    thrust::complex<double>* dev_UTermEnd;

    double* dev_HXYMid;

    hipMalloc( (void**)&dev_nPrime, numberOfThreads * ( 4 + ANCILLA_MODES ) * sizeof(int) );

    hipMalloc( (void**)&dev_mPrime, numberOfThreads * ( 2 + ANCILLA_PHOTONS ) * sizeof(int) );

    hipMalloc( (void**)&dev_UTermBegin, 4 * numberOfThreads * sizeof( thrust::complex<double> ) );

    hipMalloc( (void**)&dev_UTermEnd, 4 * numberOfThreads * sizeof( thrust::complex<double> ) );

    hipMalloc( (void**)&dev_HXYMid, numberOfThreads * sizeof(double) );

    hipMemcpy( dev_nPrime, nPrimeStarter, numberOfThreads * ( 4 + ANCILLA_MODES ) * sizeof(int), hipMemcpyHostToDevice );

    hipMemcpy( dev_mPrime, mPrimeStarter, numberOfThreads * ( 2 + ANCILLA_PHOTONS ) * sizeof(int), hipMemcpyHostToDevice );

    kernel<<<blocksPerGrid,threadsPerBlock>>>(dev_nPrime,dev_mPrime,dev_UTermBegin,dev_UTermEnd,dev_HXYMid);

    hipFree( dev_nPrime );

    hipFree( dev_mPrime );

    int* dev_reduceGridStart;
    int* dev_reduceGridEnd;

    hipMalloc( (void**)&dev_reduceGridStart, reduceGridSize * sizeof(int) );
    hipMalloc( (void**)&dev_reduceGridEnd, reduceGridSize * sizeof(int) );

    hipMemcpy( dev_reduceGridStart,reduceGridStart,reduceGridSize * sizeof(int),hipMemcpyHostToDevice );
    hipMemcpy( dev_reduceGridEnd,reduceGridEnd,reduceGridSize * sizeof(int),hipMemcpyHostToDevice );

    reduce<<<blocksPerGrid,threadsPerBlock>>>(dev_UTermBegin,dev_UTermEnd,dev_HXYMid,dev_reduceGridStart,dev_reduceGridEnd);

    hipFree( dev_reduceGridStart );
    hipFree( dev_reduceGridEnd );

    hipFree( dev_UTermBegin );

    hipFree( dev_UTermEnd );

    hipFree( dev_HXYMid );

    std::cout << "End." << std::endl;

    std::cout << "CUDA Errors: " << hipGetErrorString( hipGetLastError() ) << std::endl;

    return 1.0;

}

void CUDAOffloader::initializeStartingNPrimeMPrime(std::vector< std::vector<int> >& nPrime,std::vector< std::vector<int> >& mPrime){

    nPrimeStarter = new int[ numberOfThreads * (4 + ANCILLA_MODES) ];
    mPrimeStarter = new int[ numberOfThreads * (2 + ANCILLA_PHOTONS) ];

    gccCompiledFunctions.initializeStartingNPrimeMPrime(nPrime,mPrime,nPrimeStarter,mPrimeStarter,numberOfThreads,termIntervals);

    return;

}

void CUDAOffloader::allocateResources(){

    int count;

    hipGetDeviceCount( &count );

    assert( count > 0 );

    hipDeviceProp_t prop;

    hipGetDeviceProperties( &prop,0 );

    int spaceAvail = prop.totalGlobalMem;

    int UStorageSize = ( ANCILLA_MODES + 4 ) * ( ANCILLA_MODES + 4 ) * 2 * sizeof(double);
    int factorialStorageSize = ( ANCILLA_PHOTONS + 3 ) * sizeof(double);

    spaceAvail -= UStorageSize;
    spaceAvail -= factorialStorageSize;

    std::cout << "Space available on GPU: " << spaceAvail << " bytes" <<std::endl << std::endl;

    numberOfThreads = 0;
    int spaceTaken = 0;

    while( spaceTaken < spaceAvail ){

        numberOfThreads++;

        spaceTaken = sizeof(int) * numberOfThreads * ( 2 + 4 + ANCILLA_PHOTONS + ANCILLA_MODES );

        spaceTaken += 4 * 2 * sizeof(thrust::complex<double>) * numberOfThreads;

        spaceTaken += sizeof(double) * numberOfThreads;

    }

    numberOfThreads--;

    spaceTaken = sizeof(int) * numberOfThreads * ( 2 + 4 + ANCILLA_PHOTONS + ANCILLA_MODES );

    spaceTaken += 4 * 2 * sizeof(thrust::complex<double>) * numberOfThreads;

    spaceTaken += sizeof(double) * numberOfThreads;

    threadsPerBlock = 1024;

    std::cout << "Number of total terms: " << numberOfTerms << std::endl;
    std::cout << "Max Number of threads: " << numberOfThreads << std::endl;
    std::cout << "Space used on GPU for this number: " << spaceTaken << " bytes" << std::endl;

    termIntervals = ( numberOfTerms + numberOfThreads - 1 ) / numberOfThreads;
    termIntervals += TERMS_BUFFER;

    std::cout << "The Minimum Number of Terms that need to be evaluated in at least one interval: " << termIntervals << std::endl;

    numberOfThreads = ( numberOfTerms + termIntervals - 1 ) / termIntervals;

    std::cout << "Adjusted number of threads: " << numberOfThreads << std::endl;
    std::cout << "Number of total terms covered if each thread does " << termIntervals << " terms: "  << numberOfThreads * termIntervals << std::endl;

    while( numberOfThreads % threadsPerBlock != 0 ) threadsPerBlock--;

    blocksPerGrid = numberOfThreads / threadsPerBlock;

    std::cout << "Adjusted threads per block: " << threadsPerBlock << std::endl;
    std::cout << "Adjusted blocks per grid: " << blocksPerGrid << std::endl;

    spaceTaken = sizeof(int) * numberOfThreads * ( 2 + 4 + ANCILLA_PHOTONS + ANCILLA_MODES );

    spaceTaken += 4 * 2 * sizeof(thrust::complex<double>) * numberOfThreads;

    spaceTaken += sizeof(double) * numberOfThreads;

    std::cout << "Adjusted Space used on GPU: " << spaceTaken << " bytes" << std::endl;

    assert( threadsPerBlock * blocksPerGrid == numberOfThreads );

    hipMemcpyToSymbol(HIP_SYMBOL( dev_termIntervals),&termIntervals, sizeof(int) );

    return;

}

void CUDAOffloader::sendUToGPU(Eigen::MatrixXcd& U){

    double UArr[ 2 * (ANCILLA_MODES + 4) * (ANCILLA_MODES + 4) ];

    int k=0;

    for(int j=0;j<ANCILLA_MODES+4;j++) for(int i=0;i<ANCILLA_MODES+4;i++){

        UArr[k] = std::real( U(i,j) );
        k++;

        UArr[k] = std::imag( U(i,j) );
        k++;

    }

    hipMemcpyToSymbol(HIP_SYMBOL( dev_U),UArr, 2 * (ANCILLA_MODES + 4) * (ANCILLA_MODES + 4) * sizeof(double) );

    return;

}

CUDAOffloader::CUDAOffloader(){


}


void CUDAOffloader::setGPUDevice(int deviceNumb){

    hipSetDevice(deviceNumb);

    return;

}

void CUDAOffloader::sendFactorialToGPU(std::vector<double>& factorial){

    assert( ANCILLA_PHOTONS + 2 + 1 == factorial.size() );

    double factorialArr[factorial.size()];

    for(int i=0;i<factorial.size();i++) factorialArr[i] = factorial.at(i);

    hipMemcpyToSymbol(HIP_SYMBOL( dev_factorial),factorialArr, factorial.size() * sizeof(double) );

    return;

}


void CUDAOffloader::queryGPUDevices(){

    int count;

    hipGetDeviceCount( &count );

    std::cout << "Number of devices: " << count << std::endl << std::endl;

    for(int i=0;i<count;i++){

        hipDeviceProp_t prop;

        hipGetDeviceProperties( &prop,i );

        std::cout << "Device No. " << i << ": " << std::endl;
        std::cout << "\t" << prop.name << std::endl;
        if(prop.integrated) std::cout << "\tIntegrated GPU" << std::endl;
        else std::cout << "\tNon-integrated GPU" << std::endl;
        std::cout << "\t" << "Device compute capability: " << prop.major << "." << prop.minor << " (1.3 or higher supports double-precision math)" << std::endl;
        std::cout << "\t" << prop.totalGlobalMem << " bytes of global memory" << std::endl;
        std::cout << "\t" << prop.sharedMemPerBlock << " bytes of shared memory for a single block" << std::endl;
        std::cout << "\t" << prop.regsPerBlock << " registers (32 bit) available per block" << std::endl;
        std::cout << "\t" << prop.warpSize << " threads in a warp" << std::endl;
        std::cout << "\t" << prop.memPitch << " bytes maximum pitch allowed for memory copies" << std::endl;
        std::cout << "\t" << prop.maxThreadsPerBlock << " maximum number of threads that a block may contain" << std::endl;
        std::cout << "\t" << prop.maxThreadsDim[0] << " maximum number of threads along X" << std::endl;
        std::cout << "\t" << prop.totalConstMem << " amount of available constant memory" << std::endl;
        if(prop.deviceOverlap) std::cout << "\t" << "Device can simultaneously perform hipMemcpy() and a kernel execution" << std::endl;
        else std::cout << "\t" << "Device cannot simultaneously perform hipMemcpy() and a kernel execution" << std::endl;
        std::cout << "\t" << prop.multiProcessorCount << " multiprocessors on the device" << std::endl;
        if(prop.kernelExecTimeoutEnabled) std::cout << "\tRuntime limit for kernels on this device is enabled" << std::endl;
        else std::cout << "\tRuntime limit for kernels on this device is disabled" << std::endl;

        std::cout << "\t" << "Compute mode: " << prop.computeMode << std::endl;
        std::cout << "\t" << "Concurrent Kernels: " << prop.concurrentKernels << std::endl;

        prop.computeMode = 1;

        std::cout << std::endl;

    }

    return;

}


__device__ inline void iter_swap(int* __a, int* __b) {
  int __tmp = *__a;
  *__a = *__b;
  *__b = __tmp;
}


__device__ void reverse(int* __first, int* __last) {

  while (true)
    if (__first == __last || __first == --__last)
      return;
    else{
      iter_swap(__first++, __last);
    }
}


__device__ bool next_permutation(int* __first, int* __last) {

  if (__first == __last)
    return false;
  int* __i = __first;
  ++__i;
  if (__i == __last)
    return false;
  __i = __last;
  --__i;

  for(;;) {
    int* __ii = __i;
    --__i;
    if (*__i < *__ii) {
      int* __j = __last;
      while (!(*__i < *--__j))
        {}
    iter_swap(__i, __j);
      reverse(__ii, __last);
      return true;
    }
    if (__i == __first) {
      reverse(__first, __last);
      return false;
    }
  }

}


__device__ bool iterateNPrime(int* __begin,int* __end){

    int* ptr = __end - 2;

    while( *ptr == 0 ){

        if( ptr == __begin ) return false;

        ptr--;

    }

    *ptr -= 1;

    *( ptr + 1 ) = *( __end -1 ) + 1;

    if( ptr + 1 != __end - 1 ) *( __end - 1 ) = 0;

    return true;

}

__device__ void setMPrime( int* __nBegin, int* __mBegin ){

    int k=0;

    for(int i=0;i<ANCILLA_MODES+4;i++) for(int j=0;j < *(__nBegin + i);j++){

            *( __mBegin + k ) = i;

            k++;

    }

    return;

}
