#include "CUDAFuncs.h"

#define ANCILLA_PHOTONS 6
#define ANCILLA_MODES 8
#define HILBERT_SPACE_DIMENSION 75582

#define TERMS_BUFFER 0

__constant__ double dev_factorial[ ANCILLA_PHOTONS + 2 + 1 ];
__constant__ double dev_U[ 2 * (ANCILLA_MODES + 4) * (ANCILLA_MODES + 4) ];
thrust::complex<double>* dev_UTerms;


void CUDAOffloader::initializeStartingNPrimeMPrime(std::vector< std::vector<int> >& nPrime,std::vector< std::vector<int> >& mPrime){



    return;

}

void CUDAOffloader::allocateResources(){

    int count;

    hipGetDeviceCount( &count );

    assert( count > 0 );

    hipDeviceProp_t prop;

    hipGetDeviceProperties( &prop,0 );

    int spaceAvail = prop.totalGlobalMem;

    int UStorageSize = ( ANCILLA_MODES + 4 ) * ( ANCILLA_MODES + 4 ) * 16;
    int factorialStorageSize = ( ANCILLA_PHOTONS + 3 ) * 8;
    int UTermStorageSize = 16 * HILBERT_SPACE_DIMENSION;

    spaceAvail -= UStorageSize;
    spaceAvail -= factorialStorageSize;
    spaceAvail -= UTermStorageSize;

    std::cout << "Space available on GPU: " << spaceAvail << " bytes" <<std::endl << std::endl;

    numberOfThreads = 0;
    int spaceTaken = 0;

    while( spaceTaken < spaceAvail ){

        numberOfThreads++;

        spaceTaken = 4 * numberOfThreads * ( 2 + 4 + ANCILLA_PHOTONS + ANCILLA_MODES );

        spaceTaken += 16 * numberOfThreads;

    }

    numberOfThreads--;

    threadsPerBlock = 1024;

    std::cout << "Number of total terms: " << numberOfTerms << std::endl;
    std::cout << "Max Number of threads: " << numberOfThreads << std::endl;
    std::cout << "Space used on GPU for this number: " << 4 * numberOfThreads * ( 2 + 4 + ANCILLA_PHOTONS + ANCILLA_MODES + 4 ) << " bytes" << std::endl;

    termIntervals = ( numberOfTerms + numberOfThreads - 1 ) / numberOfThreads;
    termIntervals += TERMS_BUFFER;

    std::cout << "The Minimum Number of Terms that need to be evaluated in at least one interval: " << termIntervals << std::endl;

    numberOfThreads = ( numberOfTerms + termIntervals - 1 ) / termIntervals;

    std::cout << "Adjusted number of threads: " << numberOfThreads << std::endl;
    std::cout << "Number of total terms covered if each thread does " << termIntervals << " terms: "  << numberOfThreads * termIntervals << std::endl;

    while( numberOfThreads % threadsPerBlock != 0 ) threadsPerBlock--;

    blocksPerGrid = numberOfThreads / threadsPerBlock;

    std::cout << "Adjusted threads per block: " << threadsPerBlock << std::endl;
    std::cout << "Adjusted blocks per grid: " << blocksPerGrid << std::endl;

    std::cout << "Adjusted Space used on GPU: " << 4 * numberOfThreads * ( 2 + 4 + ANCILLA_PHOTONS + ANCILLA_MODES + 4 ) << " bytes" << std::endl;

    assert( threadsPerBlock * blocksPerGrid == numberOfThreads );

    return;

}

void CUDAOffloader::sendUToGPU(Eigen::MatrixXcd& U){

    double UArr[ 2 * (ANCILLA_MODES + 4) * (ANCILLA_MODES + 4) ];

    int k=0;

    for(int j=0;j<ANCILLA_MODES+4;j++) for(int i=0;i<ANCILLA_MODES+4;i++){

        UArr[k] = std::real( U(i,j) );
        k++;

        UArr[k] = std::imag( U(i,j) );
        k++;

    }

    hipMemcpyToSymbol(HIP_SYMBOL( dev_U),UArr, 2 * (ANCILLA_MODES + 4) * (ANCILLA_MODES + 4) * sizeof(double) );

    return;

}

CUDAOffloader::CUDAOffloader(){


}


void CUDAOffloader::setGPUDevice(int deviceNumb){

    hipSetDevice(deviceNumb);

    return;

}

void CUDAOffloader::sendFactorialToGPU(std::vector<double>& factorial){

    assert( ANCILLA_PHOTONS + 2 + 1 == factorial.size() );

    double factorialArr[factorial.size()];

    for(int i=0;i<factorial.size();i++) factorialArr[i] = factorial.at(i);

    hipMemcpyToSymbol(HIP_SYMBOL( dev_factorial),factorialArr, factorial.size() * sizeof(double) );

    return;

}


void CUDAOffloader::queryGPUDevices(){

    int count;

    hipGetDeviceCount( &count );

    std::cout << "Number of devices: " << count << std::endl << std::endl;

    for(int i=0;i<count;i++){

        hipDeviceProp_t prop;

        hipGetDeviceProperties( &prop,i );

        std::cout << "Device No. " << i << ": " << std::endl;
        std::cout << "\t" << prop.name << std::endl;
        if(prop.integrated) std::cout << "\tIntegrated GPU" << std::endl;
        else std::cout << "\tNon-integrated GPU" << std::endl;
        std::cout << "\t" << "Device compute capability: " << prop.major << "." << prop.minor << " (1.3 or higher supports double-precision math)" << std::endl;
        std::cout << "\t" << prop.totalGlobalMem << " bytes of global memory" << std::endl;
        std::cout << "\t" << prop.sharedMemPerBlock << " bytes of shared memory for a single block" << std::endl;
        std::cout << "\t" << prop.regsPerBlock << " registers (32 bit) available per block" << std::endl;
        std::cout << "\t" << prop.warpSize << " threads in a warp" << std::endl;
        std::cout << "\t" << prop.memPitch << " bytes maximum pitch allowed for memory copies" << std::endl;
        std::cout << "\t" << prop.maxThreadsPerBlock << " maximum number of threads that a block may contain" << std::endl;
        std::cout << "\t" << prop.maxThreadsDim[0] << " maximum number of threads along X" << std::endl;
        std::cout << "\t" << prop.totalConstMem << " amount of available constant memory" << std::endl;
        if(prop.deviceOverlap) std::cout << "\t" << "Device can simultaneously perform hipMemcpy() and a kernel execution" << std::endl;
        else std::cout << "\t" << "Device cannot simultaneously perform hipMemcpy() and a kernel execution" << std::endl;
        std::cout << "\t" << prop.multiProcessorCount << " multiprocessors on the device" << std::endl;
        if(prop.kernelExecTimeoutEnabled) std::cout << "\tRuntime limit for kernels on this device is enabled" << std::endl;
        else std::cout << "\tRuntime limit for kernels on this device is disabled" << std::endl;

        std::cout << "\t" << "Compute mode: " << prop.computeMode << std::endl;
        std::cout << "\t" << "Concurrent Kernels: " << prop.concurrentKernels << std::endl;

        prop.computeMode = 1;

        std::cout << std::endl;

    }

    return;

}
